// -----------------------------------------------------------------------------
//
//! \file nvidiaParticles.cu
//
//! \brief contains the nvidiaParticles class methods
//! Extension of the Class Particles for GPU
//
// -----------------------------------------------------------------------------

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/gather.h>


#include "Patch.h"
#include "gpu.h"
#include "nvidiaParticles.h"

// TODO(Etienne M): The makefile does not recognise this file and doesn't compute
// it's dependencies. If you make a modification in one of the header this file
// includes, you must `touch` this file. IF you dont do that you'll have ABI/ODR
// issues (!).

// Language: "in cell" means the number of cells for that, conversely, in cluster means 
// the number of clusters as a unit of length, etc.

////////////////////////////////////////////////////////////////////////////////
// Cell key manipulation functor definition
////////////////////////////////////////////////////////////////////////////////

//! Predicate for cell_keys
//! Return True if the entry is equal to `code`
template<int code>
struct cellKeyEquals
{
    constexpr __host__ __device__ bool
    operator()( const int& x ) const
    {
        return x == code;
    }
};

template<int key>
struct cellKeyBelow
{
    constexpr __host__ __device__ bool
    operator()( const int& x ) const
    {
        return x < key;
    }
};

namespace detail {

    ////////////////////////////////////////////////////////////////////////////////
    // Cluster manipulation functor definition
    ////////////////////////////////////////////////////////////////////////////////

    //! Cluster manipulation functionalities common to all dimension.
    //! NOTE: This only focus on GPU data manipulation. The host data shall
    //! not be handled here !
    //!
    struct Cluster
    {
    public:
        //! Same type as what is used in nvidia_cell_keys_
        //!
        using IDType         = int;
        using SizeType       = unsigned int;
        using DifferenceType = int;

    public:
        //! Compute the cell key for all the particles (not only a subset).
        //!
        static inline void
        computeParticleClusterKey( nvidiaParticles& particle_container,
                                   const Params&    parameters,
                                   const Patch&     a_parent_patch );

        //! precondition:
        //!     - nvidia_cell_keys_ shall be sorted in non decreasing order
        //!     - last_index.data() is a pointer mapped to GPU via
        //!       HostDeviceMemoryManagement
        //!
        static inline void
        computeBinIndex( nvidiaParticles& particle_container );

        //! Sorting by cluster and binning
        //!
        //! precondition:
        //!     - particle_container is already sorted by cluster or
        //!       particle_container is not sorted anymore (after a push) but
        //!       still contains the old cluster key untouched.
        //!       PartBoundCond::apply will set the keys to zero !
        //!
        static inline void
        importAndSortParticles( nvidiaParticles& particle_container,
                                nvidiaParticles& particle_to_inject,
                                const Params&    parameters,
                                const Patch&     a_parent_patch );

    protected:
        template <typename InputIterator,
                  typename ClusterType>
        static void
        doComputeParticleClusterKey( InputIterator first,
                                     InputIterator last,
                                     ClusterType   cluster_type );

    };


    template <Cluster::DifferenceType kClusterWidth>
    struct Cluster2D : public Cluster
    {
    public:
    public:
        Cluster2D( double   inverse_x_cell_dimension,
                   double   inverse_y_cell_dimension,
                   SizeType local_x_dimension_in_cell,
                   SizeType local_y_dimension_in_cell,
                   int CellStartingGlobalIndex_for_x,
                   int CellStartingGlobalIndex_for_y);

        //! Compute the cell key of a_particle. a_particle shall be a tuple (from a
        //! zipiterator).
        //! The first value of a_particle is the cell key value, the other values are
        //! the positions x and y.
        //!
        template <typename Tuple>
        __host__ __device__ IDType
        Index( const Tuple& a_particle ) const;

        //! Compute the cell key of a particle range.
        //!
        static void
        computeParticleClusterKey( nvidiaParticles& particle_container,
                                   const Params&    parameters,
                                   const Patch&     a_parent_patch );

    public:
        double   inverse_of_x_cell_dimension_;
        double   inverse_of_y_cell_dimension_;
        SizeType local_y_dimension_in_cluster_;
        int CellStartingGlobalIndex_for_x_;
        int CellStartingGlobalIndex_for_y_;
    };

    template <Cluster::DifferenceType kClusterWidth>
    struct Cluster3D : public Cluster
    {
    public:
    public:
        Cluster3D( double   inverse_x_cell_dimension,
                   double   inverse_y_cell_dimension,
                   double   inverse_z_cell_dimension,
                   SizeType local_x_dimension_in_cell,
                   SizeType local_y_dimension_in_cell,
                   SizeType local_z_dimension_in_cell,
                   int CellStartingGlobalIndex_for_x,
                   int CellStartingGlobalIndex_for_y,
                   int CellStartingGlobalIndex_for_z);

        //! Compute the cell key of a_particle. a_particle shall be a tuple (from a
        //! zipiterator).
        //! The first value of a_particle is the cell key value, the other values are
        //! the positions x and y.
        //!
        template <typename Tuple>
        __host__ __device__ IDType
        Index( const Tuple& a_particle ) const;

        //! Compute the cell key of a particle range.
        //!
        static void
        computeParticleClusterKey( nvidiaParticles& particle_container,
                                   const Params&    parameters,
                                   const Patch&     a_parent_patch );

    public:
        double   inverse_of_x_cell_dimension_;
        double   inverse_of_y_cell_dimension_;
        double   inverse_of_z_cell_dimension_;
        SizeType local_y_dimension_in_cluster_;
        SizeType local_z_dimension_in_cluster_;
        int CellStartingGlobalIndex_for_x_;
        int CellStartingGlobalIndex_for_y_;
        int CellStartingGlobalIndex_for_z_;
    };


    //! This functor assign a cluster key to a_particle.
    //!
    template <typename ClusterType>
    class AssignClusterIndex
    {
    public:
    public:
        AssignClusterIndex( ClusterType cluster_type )
            : cluster_type_{ cluster_type }
        {
            // EMPTY
        }

        template <typename Tuple>
        __host__ __device__ void
        operator()( Tuple& a_particle ) const
        {
            thrust::get<0>( a_particle ) /* cluster key */ = cluster_type_.Index( a_particle );
        }

    protected:
        ClusterType cluster_type_;
    };


    ////////////////////////////////////////////////////////////////////////////////
    // Cluster manipulation functor method definitions
    ////////////////////////////////////////////////////////////////////////////////

    inline void
    Cluster::computeParticleClusterKey( nvidiaParticles& particle_container,
                                        const Params&    parameters,
                                        const Patch&     a_parent_patch )
    {
        // This is where we do a runtime dispatch depending on the simulation's
        // dimensions.

        switch( particle_container.dimension() ) {
            case 2: {
                Cluster2D<Params::getGPUClusterWidth( 2 )>::computeParticleClusterKey( particle_container,
                                                                                                parameters,
                                                                                                a_parent_patch );
                break;
            }
            case 3: {
                Cluster3D<Params::getGPUClusterWidth( 3 )>::computeParticleClusterKey( particle_container,
                                                                                                parameters,
                                                                                                a_parent_patch );
                break;
            }
            default:
                // Not implemented, only Cartesian 2D or 3D for the moment
                SMILEI_ASSERT( false );
                break;
        }
    }

    inline void
    Cluster::computeBinIndex( nvidiaParticles& particle_container )
    {
        SMILEI_GPU_ASSERT_MEMORY_IS_ON_DEVICE( particle_container.last_index.data() );

        Cluster::IDType* bin_upper_bound = smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( particle_container.last_index.data() );

        // SMILEI_ASSERT( thrust::is_sorted( thrust::device,
        //                                   static_cast<const IDType*>( particle_container.getPtrCellKeys() ),
        //                                   static_cast<const IDType*>( particle_container.getPtrCellKeys() ) + particle_container.deviceSize() ) );

        // NOTE: On some benchmark, I found this upper_bound usage faster than the counting_iterator (by a lot(!) ~x3, but
        // it's so fast anyway..)

        // thrust::upper_bound( thrust::device,
        //                      nvidia_cell_keys_.cbegin(), nvidia_cell_keys_.cend(),
        //                      key_bound_to_search.cbegin(), key_bound_to_search.cend(),
        //                      bin_upper_bound );

        // NOTE: A particle is in a bin if the index of the bin is the same integer value as the particle's cell key.
        // The particles are sorted by cell key. We can do a simple binary search to find the upper bound of a bin.
        //
        thrust::upper_bound( thrust::device,
                             static_cast<const IDType*>( particle_container.getPtrCellKeys() ),
                             static_cast<const IDType*>( particle_container.getPtrCellKeys() ) + particle_container.deviceSize(),
                             thrust::counting_iterator<Cluster::IDType>{ static_cast<Cluster::IDType>( 0 ) },
                             thrust::counting_iterator<Cluster::IDType>{ static_cast<Cluster::IDType>( particle_container.last_index.size() ) },
                             bin_upper_bound );

        // SMILEI_ASSERT( thrust::is_sorted( thrust::device,
        //                                   bin_upper_bound,
        //                                   bin_upper_bound + particle_container.last_index.size() ) );
    }

    inline void
    Cluster::importAndSortParticles( nvidiaParticles& particle_container,
                                     nvidiaParticles& particle_to_inject,
                                     const Params&    parameters,
                                     const Patch&     a_parent_patch )
    {
        // Remove out of bound particles
        const auto erased_count = particle_container.eraseParticlesByPredicate( cellKeyBelow<0>() );
        
        const auto initial_count = particle_container.deviceSize() - erased_count;
        const auto inject_count  = particle_to_inject.deviceSize();
        const auto new_count     = initial_count + inject_count;
        
        // Resize particles
        // NOTE: We really want a non-initializing vector here!
        // It's possible to give a custom allocator to thrust::device_vector.
        // Create one with construct(<>) as a noop and derive from
        // thrust::device_malloc_allocator. For now we do an explicit resize.
        particle_container.softReserve( new_count );
        particle_container.resize( new_count );
        
        // Combine imported particles to main particles
        particle_container.copyParticles( &particle_to_inject, initial_count );
        
        // Compute keys of particles
        computeParticleClusterKey( particle_container, parameters, a_parent_patch );
        
        // Use particle_to_inject as a buffer
        particle_to_inject.softReserve( new_count );
        particle_to_inject.resize( new_count );
        
        // Sort particles using thrust::gather, according to the sorting map
        particle_container.sortParticleByKey( particle_to_inject );
        
        // Recompute bins
        computeBinIndex( particle_container );
        
        // This free generates a lot of memory fragmentation. If we enable it we
        // reduce significantly the memory usage over time but a memory spike
        // will still be present. Unfortunately, this free generates soo much
        // fragmentation (like the one above) that at some point the GPU memory
        // allocator will fail!
        // particle_to_inject.free();
    }

    template <typename InputIterator,
              typename ClusterType>
    void
    Cluster::doComputeParticleClusterKey( InputIterator first,
                                          InputIterator last,
                                          ClusterType   cluster_type )
    {
        thrust::for_each( thrust::device,
                          first, last,
                          AssignClusterIndex<ClusterType>{ cluster_type } );
    }

    ////////////////////////////////////////////////////////////////////////////////
    // Cluster2D method definitions
    ////////////////////////////////////////////////////////////////////////////////

    template <Cluster::DifferenceType kClusterWidth>
    Cluster2D<kClusterWidth>::Cluster2D( double   inverse_x_cell_dimension,
                                         double   inverse_y_cell_dimension,
                                         SizeType local_x_dimension_in_cell,
                                         SizeType local_y_dimension_in_cell,
                                         int CellStartingGlobalIndex_for_x, int CellStartingGlobalIndex_for_y )
        : inverse_of_x_cell_dimension_{ inverse_x_cell_dimension }
        , inverse_of_y_cell_dimension_{ inverse_y_cell_dimension }
        , local_y_dimension_in_cluster_{ local_y_dimension_in_cell / kClusterWidth }
        , CellStartingGlobalIndex_for_x_{CellStartingGlobalIndex_for_x}
        , CellStartingGlobalIndex_for_y_{CellStartingGlobalIndex_for_y}
    {
        // EMPTY
    }

    template <Cluster::DifferenceType kClusterWidth>
    Cluster3D<kClusterWidth>::Cluster3D( double   inverse_x_cell_dimension,
                                         double   inverse_y_cell_dimension,
                                         double   inverse_z_cell_dimension,
                                         SizeType local_x_dimension_in_cell,
                                         SizeType local_y_dimension_in_cell,
                                         SizeType local_z_dimension_in_cell,
                                         int CellStartingGlobalIndex_for_x,
                                         int CellStartingGlobalIndex_for_y, int CellStartingGlobalIndex_for_z )
        : inverse_of_x_cell_dimension_{ inverse_x_cell_dimension }
        , inverse_of_y_cell_dimension_{ inverse_y_cell_dimension }
        , inverse_of_z_cell_dimension_{ inverse_z_cell_dimension }
        , local_y_dimension_in_cluster_{ local_y_dimension_in_cell / kClusterWidth }
        , local_z_dimension_in_cluster_{ local_z_dimension_in_cell / kClusterWidth }
        , CellStartingGlobalIndex_for_x_{CellStartingGlobalIndex_for_x}
        , CellStartingGlobalIndex_for_y_{CellStartingGlobalIndex_for_y}
        , CellStartingGlobalIndex_for_z_{CellStartingGlobalIndex_for_z}
    {
        // EMPTY
    }

    template <Cluster::DifferenceType kClusterWidth>
    template <typename Tuple>
    __host__ __device__ typename Cluster2D<kClusterWidth>::IDType
    Cluster2D<kClusterWidth>::Index( const Tuple& a_particle ) const
    {
        const SizeType local_x_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<1>( a_particle ) *
                                                                                    inverse_of_x_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_x_;
        const SizeType local_y_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<2>( a_particle ) *
                                                                                    inverse_of_y_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_y_;

        // These divisions will be optimized.
        // The integer division rounding behavior is expected.

        // NOTE: Flat tiles have been studied but were not as efficient for the
        // projection. The square provides the minimal perimeter (and thus ghost
        // cell amount) for a given area.
        static constexpr SizeType x_cluster_dimension_in_cell = kClusterWidth;
        static constexpr SizeType y_cluster_dimension_in_cell = kClusterWidth;

        const SizeType local_x_particle_cluster_coordinate_in_cluster = local_x_particle_coordinate_in_cell / x_cluster_dimension_in_cell;
        const SizeType local_y_particle_cluster_coordinate_in_cluster = local_y_particle_coordinate_in_cell / y_cluster_dimension_in_cell;

        const SizeType y_stride = local_y_dimension_in_cluster_;

        // The indexing order is: x * ywidth * zwidth + y * zwidth + z
        const SizeType cluster_index = local_x_particle_cluster_coordinate_in_cluster * y_stride +
                                       local_y_particle_cluster_coordinate_in_cluster;

        return static_cast<IDType>( cluster_index );
    }
    
    template <Cluster::DifferenceType kClusterWidth>
    template <typename Tuple>
    __host__ __device__ typename Cluster3D<kClusterWidth>::IDType
    Cluster3D<kClusterWidth>::Index( const Tuple& a_particle ) const
    {
        const SizeType local_x_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<1>( a_particle ) *
                                                                                    inverse_of_x_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_x_;
        const SizeType local_y_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<2>( a_particle ) *
                                                                                    inverse_of_y_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_y_;
        const SizeType local_z_particle_coordinate_in_cell = static_cast<SizeType>( thrust::get<3>( a_particle ) *
                                                                                    inverse_of_z_cell_dimension_ ) -
                                                             CellStartingGlobalIndex_for_z_;

        // These divisions will be optimized.
        // The integer division rounding behavior is expected.

        // NOTE: Flat tiles have been studied but were not as efficient for the
        // projection. The square provides the minimal perimeter (and thus ghost
        // cell amount) for a given area.
        static constexpr SizeType x_cluster_dimension_in_cell = kClusterWidth;
        static constexpr SizeType y_cluster_dimension_in_cell = kClusterWidth;
        static constexpr SizeType z_cluster_dimension_in_cell = kClusterWidth;

        const SizeType local_x_particle_cluster_coordinate_in_cluster = local_x_particle_coordinate_in_cell / x_cluster_dimension_in_cell;
        const SizeType local_y_particle_cluster_coordinate_in_cluster = local_y_particle_coordinate_in_cell / y_cluster_dimension_in_cell;
        const SizeType local_z_particle_cluster_coordinate_in_cluster = local_z_particle_coordinate_in_cell / z_cluster_dimension_in_cell;

        const SizeType y_stride = local_y_dimension_in_cluster_;
        const SizeType z_stride = local_z_dimension_in_cluster_;

        // The indexing order is: x * ywidth * zwidth + y * zwidth + z
        const SizeType cluster_index = local_x_particle_cluster_coordinate_in_cluster * z_stride * y_stride +
                                       local_y_particle_cluster_coordinate_in_cluster * z_stride +
                                       local_z_particle_cluster_coordinate_in_cluster;

        return static_cast<IDType>( cluster_index );
    }

    template <Cluster::DifferenceType kClusterWidth>
    void
    Cluster2D<kClusterWidth>::computeParticleClusterKey( nvidiaParticles& particle_container,
                                                         const Params&    parameters,
                                                         const Patch&     a_parent_patch )
    {
        const auto first = thrust::make_zip_iterator( thrust::make_tuple( particle_container.getPtrCellKeys(),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 0 ) ),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 1 ) ) ) );
        const auto last  = first + particle_container.deviceSize();
        int CellStartingGlobalIndex_for_x = a_parent_patch.getCellStartingGlobalIndex_noGC(0);
        int CellStartingGlobalIndex_for_y = a_parent_patch.getCellStartingGlobalIndex_noGC(1);
        doComputeParticleClusterKey( first, last,
                                     Cluster2D<Params::getGPUClusterWidth( 2 )>{ parameters.res_space[0],
                                                                                          parameters.res_space[1],
                                                                                          parameters.patch_size_[0],
                                                                                          parameters.patch_size_[1],
                                                                                          CellStartingGlobalIndex_for_x,
                                                                                          CellStartingGlobalIndex_for_y } );
    }

    template <Cluster::DifferenceType kClusterWidth>
    void
    Cluster3D<kClusterWidth>::computeParticleClusterKey( nvidiaParticles& particle_container,
                                                         const Params&    parameters,
                                                         const Patch&     a_parent_patch )
    {
        const auto first = thrust::make_zip_iterator( thrust::make_tuple( particle_container.getPtrCellKeys(),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 0 ) ),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 1 ) ),
                                                                          static_cast<const double*>( particle_container.getPtrPosition( 2 ) ) ) );
        const auto last  = first + particle_container.deviceSize();
        int CellStartingGlobalIndex_for_x = a_parent_patch.getCellStartingGlobalIndex_noGC(0);
        int CellStartingGlobalIndex_for_y = a_parent_patch.getCellStartingGlobalIndex_noGC(1);
        int CellStartingGlobalIndex_for_z = a_parent_patch.getCellStartingGlobalIndex_noGC(2);
        doComputeParticleClusterKey( first, last,
                                     Cluster3D<Params::getGPUClusterWidth( 3 )>{ parameters.res_space[0],
                                                                                          parameters.res_space[1],
                                                                                          parameters.res_space[2],
                                                                                          parameters.patch_size_[0],
                                                                                          parameters.patch_size_[1],
                                                                                          parameters.patch_size_[2],
                                                                                          CellStartingGlobalIndex_for_x,
                                                                                          CellStartingGlobalIndex_for_y,
                                                                                          CellStartingGlobalIndex_for_z } );
    }

} // namespace detail


////////////////////////////////////////////////////////////////////////////////
// nvidiaParticles method definitions
////////////////////////////////////////////////////////////////////////////////

nvidiaParticles::nvidiaParticles( const Params& parameters,
                                  const Patch&  a_parent_patch )
    : Particles{}
    , parameters_{ &parameters }
    , parent_patch_{ &a_parent_patch }
    , gpu_nparts_{}
{
    // EMPTY
}

nvidiaParticles::~nvidiaParticles() {
    // Manage last_index if allocated on GPU
    if (smilei::tools::gpu::HostDeviceMemoryManagement::IsHostPointerMappedOnDevice( last_index.data() )) {
        smilei::tools::gpu::HostDeviceMemoryManagement::DeviceFree( last_index );
    }
}

void nvidiaParticles::resizeDimensions( unsigned int nDim )
{
    nvidia_position_.resize( nDim );
    nvidia_momentum_.resize( 3 );
}

void nvidiaParticles::softReserve( unsigned int particle_count, float growth_factor  )
{
    if( particle_count <= deviceCapacity() ) {
        // Dont reserve, for now we have enough capacity.
        return;
    }

    const unsigned int new_capacity = static_cast<unsigned int>( particle_count * growth_factor );

    for( unsigned int idim = 0; idim < nvidia_position_.size(); idim++ ) {
        nvidia_position_[idim].reserve( new_capacity );
    }

    for( unsigned int idim = 0; idim < 3; idim++ ) {
        nvidia_momentum_[idim].reserve( new_capacity );
    }

    nvidia_weight_.reserve( new_capacity );
    nvidia_charge_.reserve( new_capacity );

    if( has_quantum_parameter ) {
        nvidia_chi_.reserve( new_capacity );
    }

    if( has_Monte_Carlo_process ) {
        nvidia_tau_.reserve( new_capacity );
    }

    if( tracked ) {
        nvidia_id_.reserve( new_capacity );
    }

    nvidia_cell_keys_.reserve( new_capacity );
}

void nvidiaParticles::reserve( unsigned int particle_count )
{
    for( unsigned int idim = 0; idim < nvidia_position_.size(); idim++ ) {
        nvidia_position_[idim].reserve( particle_count );
    }

    for( unsigned int idim = 0; idim < 3; idim++ ) {
        nvidia_momentum_[idim].reserve( particle_count );
    }

    nvidia_weight_.reserve( particle_count );
    nvidia_charge_.reserve( particle_count );

    if( has_quantum_parameter ) {
        nvidia_chi_.reserve( particle_count );
    }

    if( has_Monte_Carlo_process ) {
        nvidia_tau_.reserve( particle_count );
    }

    if( tracked ) {
        nvidia_id_.reserve( particle_count );
    }

    nvidia_cell_keys_.reserve( particle_count );
}

void nvidiaParticles::resize( unsigned int particle_count )
{

    // TODO(Etienne M): Use non-initializing vector/allocator (dont pay the cost
    // of what you dont use) ?

    for( int idim = 0; idim < nvidia_position_.size(); idim++ ) {
        nvidia_position_[idim].resize( particle_count );
    }

    for( int idim = 0; idim < 3; idim++ ) {
        nvidia_momentum_[idim].resize( particle_count );
    }

    nvidia_weight_.resize( particle_count );
    nvidia_charge_.resize( particle_count );

    if( has_quantum_parameter ) {
        nvidia_chi_.resize( particle_count );
    }

    if( has_Monte_Carlo_process ) {
        nvidia_tau_.resize( particle_count );
    }

    if( tracked ) {
        nvidia_id_.resize( particle_count );
    }

    nvidia_cell_keys_.resize( particle_count );

    gpu_nparts_ = particle_count;
}

void nvidiaParticles::free()
{
    for( auto& a_vector : nvidia_position_ ) {
        thrust::device_vector<double> a_dummy_vector{};
        std::swap( a_vector, a_dummy_vector );
    }

    for( auto& a_vector : nvidia_momentum_ ) {
        thrust::device_vector<double> a_dummy_vector{};
        std::swap( a_vector, a_dummy_vector );
    }

    {
        thrust::device_vector<double> a_dummy_vector{};
        std::swap( nvidia_weight_, a_dummy_vector );
    }

    {
        thrust::device_vector<short> a_dummy_vector{};
        std::swap( nvidia_charge_, a_dummy_vector );
    }

    if( has_quantum_parameter ) {
        thrust::device_vector<double> a_dummy_vector{};
        std::swap( nvidia_chi_, a_dummy_vector );
    }

    if( has_Monte_Carlo_process ) {
        thrust::device_vector<double> a_dummy_vector{};
        std::swap( nvidia_tau_, a_dummy_vector );
    }

    if( tracked ) {
        thrust::device_vector<uint64_t> a_dummy_vector{};
        std::swap( nvidia_id_, a_dummy_vector );
    }

    {
        thrust::device_vector<int> a_dummy_vector{};
        std::swap( nvidia_cell_keys_, a_dummy_vector );
    }

    gpu_nparts_ = 0;
}

// ---------------------------------------------------------------------------------------------------------------------
//! Resize particle vectors
// ---------------------------------------------------------------------------------------------------------------------
void nvidiaParticles::deviceResize( unsigned int new_size )
{
    for( unsigned int iprop=0 ; iprop<nvidia_double_prop_.size() ; iprop++ ) {
        ( *nvidia_double_prop_[iprop] ).resize(new_size);
    }

    for( unsigned int iprop=0 ; iprop<nvidia_short_prop_.size() ; iprop++ ) {
        ( *nvidia_short_prop_[iprop] ).resize(new_size);
    }

    //
    // for( unsigned int iprop=0 ; iprop<uint64_prop.size() ; iprop++ ) {
    //     ( *nvidia_uint64_prop[iprop] ).resize( n_particles+n_additional_particles );
    // }

    if (tracked) {
        nvidia_id_.resize( new_size );
    }

    nvidia_cell_keys_.resize( new_size );

    gpu_nparts_ = new_size;
}

// ---------------------------------------------------------------------------------------------------------------------
// Reset of Particles vectors
// Cell keys not affected
// ---------------------------------------------------------------------------------------------------------------------
void nvidiaParticles::deviceClear()
{
    for( unsigned int iprop = 0; iprop < nvidia_double_prop_.size(); iprop++ ) {
        nvidia_double_prop_[iprop]->clear();
    }

    for( unsigned int iprop = 0; iprop < nvidia_short_prop_.size(); iprop++ ) {
        nvidia_short_prop_[iprop]->clear();
    }

    // TODO(Etienne M): Clear cell keys too ?

    if (tracked) {
        nvidia_id_.clear();
    }

    gpu_nparts_ = 0;
}

// ---------------------------------------------------------------------------------------------------------------------
//! Reset cell_keys to default value
// ---------------------------------------------------------------------------------------------------------------------
void nvidiaParticles::resetCellKeys(void)
{
    thrust::fill(nvidia_cell_keys_.begin(), nvidia_cell_keys_.begin() + gpu_nparts_, -1);
}

// -----------------------------------------------------------------------------
//! Initialize the particle properties on device as a mirror of the host definition
// -----------------------------------------------------------------------------
void nvidiaParticles::initializeDataOnDevice()
{
    SMILEI_ASSERT( Position.size() > 0 );
    // The world shall end if we call this function multiple times
    SMILEI_ASSERT( nvidia_double_prop_.empty() );

    const auto kPositionDimension = Position.size();

    // We sure that we have as many say, position dimension as the base class.
    resizeDimensions( kPositionDimension );

    // Initialize the list of pointers

    for( unsigned int i = 0; i < kPositionDimension; i++ ) {
        nvidia_double_prop_.push_back( &nvidia_position_[i] );
    }

    for( unsigned int i = 0; i < 3; i++ ) {
        nvidia_double_prop_.push_back( &nvidia_momentum_[i] );
    }

    nvidia_double_prop_.push_back( &nvidia_weight_ );

    nvidia_short_prop_.push_back( &nvidia_charge_ );

    // Quantum parameter (for QED effects):
    // - if radiation reaction (continuous or discontinuous)
    // - if multiphoton-Breit-Wheeler if photons
    if( has_quantum_parameter ) {
        nvidia_double_prop_.push_back( &nvidia_chi_ );
    }

    // Optical Depth for Monte-Carlo processes:
    // - if the discontinuous (Monte-Carlo) radiation reaction
    // is activated, tau is the incremental optical depth to emission
    if( has_Monte_Carlo_process ) {
        nvidia_double_prop_.push_back( &nvidia_tau_ );
    }

    const auto kHostParticleCount = Position[0].size();

    if( kHostParticleCount == 0 ) {
        // Should we reserve some space ?
        // reserve( 100 );
    } else {
        copyFromHostToDevice();
    }

    if( prepareBinIndex() < 0 ) {
        // Either we deal with a simulation with unsupported space dimensions
        // (1D/AM) or we are not using OpenMP or we are dealing with particle
        // object without allocated bin (particle_to_move for instance).
        // We'll use the old, naive, unsorted particles injection
        // implementation.

        // Dont call setHostBinIndex. For particle that have binning this is a
        // redundant call. But for the particle that should not get binned
        // (ie: particle_to_move) , this is a bug (!) and will trigger an
        // assertion.

        // setHostBinIndex();
    } else {

        // At this point, a copy of the host particles and last_index is on the
        // device and we know we support the space dimension.

        detail::Cluster::computeParticleClusterKey( *this, *parameters_, *parent_patch_ );

        // The particles are not correctly sorted when created.
        sortParticleByKey();

        detail::Cluster::computeBinIndex( *this );
        setHostBinIndex();
    }
}

// -------------------------------------------------------------------------------------------------
//! Copy particle IDs from host to device
// -------------------------------------------------------------------------------------------------
void nvidiaParticles::initializeIDsOnDevice()
{
    nvidia_id_.resize( Id.size() );
    thrust::copy((Id).begin(), (Id).end(), (nvidia_id_).begin());
}

// -------------------------------------------------------------------------------------------------
//! Copy the particles from host to device
// -------------------------------------------------------------------------------------------------
void nvidiaParticles::copyFromHostToDevice()
{
    resize( Position[0].size() );

    for( int idim = 0; idim < Position.size(); idim++ ) {
        thrust::copy( Position[idim].begin(), Position[idim].end(), nvidia_position_[idim].begin() );
    }

    for( int idim = 0; idim < Momentum.size(); idim++ ) {
        thrust::copy( Momentum[idim].begin(), Momentum[idim].end(), nvidia_momentum_[idim].begin() );
    }

    thrust::copy( Weight.begin(), Weight.end(), nvidia_weight_.begin() );

    thrust::copy( Charge.begin(), Charge.end(), nvidia_charge_.begin() );

    if( has_quantum_parameter ) {
        thrust::copy( Chi.begin(), Chi.end(), nvidia_chi_.begin() );
    }

    if( has_Monte_Carlo_process ) {
        thrust::copy( Tau.begin(), Tau.end(), nvidia_tau_.begin() );
    }

    if( tracked ) {
        thrust::copy( Id.begin(), Id.end(), nvidia_id_.begin() );
    }
}

// -------------------------------------------------------------------------------------------------
//! Copy device to host
// -------------------------------------------------------------------------------------------------
void nvidiaParticles::copyFromDeviceToHost( bool copy_keys )
{
    for (int idim=0;idim<Position.size();idim++) {
        Position[idim].resize( gpu_nparts_ );
        thrust::copy((nvidia_position_[idim]).begin(), (nvidia_position_[idim]).begin()+gpu_nparts_, (Position[idim]).begin());
    }
    for (int idim=0;idim<Momentum.size();idim++) {
        Momentum[idim].resize( gpu_nparts_ );
        thrust::copy((nvidia_momentum_[idim]).begin(), (nvidia_momentum_[idim]).begin()+gpu_nparts_, (Momentum[idim]).begin());
    }
    Weight.resize( gpu_nparts_ );
    thrust::copy((nvidia_weight_).begin(), (nvidia_weight_).begin()+gpu_nparts_, (Weight).begin());
    Charge.resize( gpu_nparts_ );
    thrust::copy((nvidia_charge_).begin(), (nvidia_charge_).begin()+gpu_nparts_, (Charge).begin());
    if (has_quantum_parameter) {
        Chi.resize( gpu_nparts_ );
        thrust::copy((nvidia_chi_).begin(), (nvidia_chi_).begin()+gpu_nparts_, (Chi).begin());
    }
    if (has_Monte_Carlo_process) {
        Tau.resize( gpu_nparts_ );
        thrust::copy((nvidia_tau_).begin(), (nvidia_tau_).begin()+gpu_nparts_, (Tau).begin());
    }
    if (tracked) {
        Id.resize( gpu_nparts_ );
        thrust::copy((nvidia_id_).begin(), (nvidia_id_).begin()+gpu_nparts_, (Id).begin());
    }
    if (copy_keys) {
        cell_keys.resize( gpu_nparts_ );
        thrust::copy((nvidia_cell_keys_).begin(), (nvidia_cell_keys_).begin()+gpu_nparts_, (cell_keys).begin());
    }
}

unsigned int nvidiaParticles::deviceCapacity() const
{
    SMILEI_ASSERT( nvidia_momentum_.size() >= 1 );
    // Could be any particle component that we know will be used in any case.
    return nvidia_momentum_[0].capacity();
}

// -----------------------------------------------------------------------------
//! Move leaving particles to the buffer
// -----------------------------------------------------------------------------
void nvidiaParticles::copyLeavingParticlesToBuffer( Particles* buffer )
{
    copyParticlesByPredicate( buffer, cellKeyBelow<-1>() );
    buffer->copyFromDeviceToHost( true );
}


//! Copy particles which statisfy some predicate
template<typename Predicate>
void nvidiaParticles::copyParticlesByPredicate( Particles* buffer, Predicate pred )
{
    // TODO(Etienne M): We are doing extra work. We could use something like
    // std::partition to output the invalidated particles in buffer
    // and keep the good ones. This would help us avoid the std::remove_if in
    // the particle injection and sorting algorithm.
    
    // Count particles satisfying the predicate
    const auto keys = getPtrCellKeys();
    const int nparts_to_copy = thrust::count_if( thrust::device, keys, keys + gpu_nparts_, pred );
    
    // Resize destination buffer (copy_if does not resize)
    nvidiaParticles* const dest = static_cast<nvidiaParticles*>( buffer );
    dest->resize( nparts_to_copy );
    
    if( nparts_to_copy ) {
        // Copy the particles to the destination
        for( int ip = 0; ip < getNDoubleProp(); ip++ ) {
            const auto in = getPtrDoubleProp( ip );
            const auto out = dest->getPtrDoubleProp( ip );
            thrust::copy_if( thrust::cuda::par_nosync, in, in + gpu_nparts_, keys, out, pred );
        }
        for( int ip = 0; ip < getNShortProp(); ip++ ) {
            const auto in = getPtrShortProp( ip );
            const auto out = dest->getPtrShortProp( ip );
            thrust::copy_if( thrust::cuda::par_nosync, in, in + gpu_nparts_, keys, out, pred );
        }
        if( tracked ) {
            const auto in = getPtrId();
            const auto out = dest->getPtrId();
            thrust::copy_if( thrust::cuda::par_nosync, in, in + gpu_nparts_, keys, out, pred );
        }
        hipDeviceSynchronize();
    }
}

void nvidiaParticles::copyParticles( Particles* particles_to_inject )
{
    const auto nparts = gpu_nparts_;
    nvidiaParticles* to_inject = static_cast<nvidiaParticles*>( particles_to_inject );
    resize( nparts + to_inject->gpu_nparts_ );
    copyParticles( to_inject, nparts );
}

void nvidiaParticles::copyParticles( nvidiaParticles* particles_to_inject, size_t offset )
{
    // Copy the particles to the destination
    for( int ip = 0; ip < getNDoubleProp(); ip++ ) {
        const auto in = particles_to_inject->getPtrDoubleProp( ip );
        const auto out = getPtrDoubleProp( ip );
        thrust::copy_n( thrust::cuda::par_nosync, in, particles_to_inject->gpu_nparts_, out + offset );
    }
    for( int ip = 0; ip < getNShortProp(); ip++ ) {
        const auto in = particles_to_inject->getPtrShortProp( ip );
        const auto out = getPtrShortProp( ip );
        thrust::copy_n( thrust::cuda::par_nosync, in, particles_to_inject->gpu_nparts_, out + offset );
    }
    if( tracked ) {
        const auto in = particles_to_inject->getPtrId();
        const auto out = getPtrId();
        thrust::copy_n( thrust::cuda::par_nosync, in, particles_to_inject->gpu_nparts_, out + offset );
    }
    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------
//! Erase `npart` particles from `ipart`
// -----------------------------------------------------------------------------
//void nvidiaParticles::eraseParticleOnDevice(int ipart, int npart) {
//
//    const auto first_particle = thrust::make_zip_iterator( thrust::make_tuple( std::begin( nvidia_position_[0] ),
//                                                                               std::begin( nvidia_momentum_[0] ),
//                                                                               std::begin( nvidia_momentum_[1] ),
//                                                                               std::begin( nvidia_momentum_[2] ),
//                                                                               std::begin( nvidia_weight_ ),
//                                                                               std::begin( nvidia_charge_ ) ) );
//
//    // Remove the other position values depending on the simulation's grid
//    // dimensions
//    for( int i = 1; i < position_dimension_count; ++i ) {
//        thrust::remove_if( thrust::device,
//                           std::begin( nvidia_position_[i] ),
//                           std::begin( nvidia_position_[i] ) + nparts,
//                           std::cbegin( nvidia_cell_keys_ ),
//                           cellKeyEquals<-1>() );
//    }
//
//}

// -----------------------------------------------------------------------------
//! Erase particles leaving the patch on device
// -----------------------------------------------------------------------------
int nvidiaParticles::eraseLeavingParticles()
{
    const auto nremoved = eraseParticlesByPredicate( cellKeyBelow<0>() );
    resize( gpu_nparts_ - nremoved );
    return nremoved;
}

//! "Erase" particles but does not resize the arrays!
template<typename Predicate>
int nvidiaParticles::eraseParticlesByPredicate( Predicate pred )
{
    const auto keys = getPtrCellKeys();
    const int nparts_to_remove = thrust::count_if( thrust::device, keys, keys + gpu_nparts_, pred );
    
    // Copy the particles to the destination
    // Using more memory, we could use the faster remove_copy_if
    // NOTE: remove_if is stable.
    for( int ip = 0; ip < getNDoubleProp(); ip++ ) {
        const auto in = getPtrDoubleProp( ip );
        thrust::remove_if( thrust::cuda::par_nosync, in, in + gpu_nparts_, keys, pred );
    }
    for( int ip = 0; ip < getNShortProp(); ip++ ) {
        const auto in = getPtrShortProp( ip );
        thrust::remove_if( thrust::cuda::par_nosync, in, in + gpu_nparts_, keys, pred );
    }
    if( tracked ) {
        const auto in = getPtrId();
        thrust::remove_if( thrust::cuda::par_nosync, in, in + gpu_nparts_, keys, pred );
    }
    hipDeviceSynchronize();
    
    return nparts_to_remove;
}


// ---------------------------------------------------------------------------------------------------------------------
//! Create n_additional_particles new particles at the end of vectors
//! Fill the new elements with 0
// ---------------------------------------------------------------------------------------------------------------------
void nvidiaParticles::createParticles( int n_additional_particles )
{
    int n_particles = gpu_nparts_;
    int new_size = n_particles + n_additional_particles;
    for( unsigned int iprop=0 ; iprop<nvidia_double_prop_.size() ; iprop++ ) {
        ( *nvidia_double_prop_[iprop] ).resize(new_size);
         thrust::fill(( *nvidia_double_prop_[iprop] ).begin() + n_particles, ( *nvidia_double_prop_[iprop] ).begin() + new_size, 0);
    }

    for( unsigned int iprop=0 ; iprop<nvidia_short_prop_.size() ; iprop++ ) {
        ( *nvidia_short_prop_[iprop] ).resize(new_size);
        thrust::fill(( *nvidia_short_prop_[iprop] ).begin() + n_particles, ( *nvidia_short_prop_[iprop] ).begin() + new_size, 0);
    }

    // for( unsigned int iprop=0 ; iprop<uint64_prop.size() ; iprop++ ) {
    //     ( *nvidia_uint64_prop[iprop] ).resize( n_particles+n_additional_particles );
    // }

    if (tracked) {
        nvidia_id_.resize( new_size );
        thrust::fill( nvidia_id_.begin() + n_particles, nvidia_id_.begin() + new_size, 0 );
    }

    nvidia_cell_keys_.resize( new_size );
    thrust::fill( nvidia_cell_keys_.begin() + n_particles, nvidia_cell_keys_.begin() + new_size, -1 );

    gpu_nparts_ = new_size;
}

//! Import Particles and sort depending if Binning is available or not
void nvidiaParticles::importAndSortParticles( Particles* particles_to_inject )
{
    if( parameters_->isGPUParticleBinningAvailable() ) {
        detail::Cluster::importAndSortParticles( *static_cast<nvidiaParticles*>( this ),
                                                 *static_cast<nvidiaParticles*>( particles_to_inject ),
                                                 *parameters_,
                                                 *parent_patch_ );
    } else {
        // When GPU particle binning is not supported, fallback to a naive implementation
        naiveImportAndSortParticles( static_cast<nvidiaParticles*>( particles_to_inject ) );
    }

    setHostBinIndex();
}

//! Sort by cell_keys_
//! This version synchronizes for every vector, but uses less buffers
void nvidiaParticles::sortParticleByKey()
{
    // Make a sorting map using the cell keys (like numpy.argsort)
    thrust::device_vector<int> index( gpu_nparts_ );
    thrust::sequence( thrust::device, index.begin(), index.end() );
    thrust::sort_by_key( thrust::device, nvidia_cell_keys_.begin(), nvidia_cell_keys_.end(), index.begin() );
    
    // Sort particles using thrust::gather, according to the sorting map
    thrust::device_vector<double> buffer( gpu_nparts_ );
    for( int ip = 0; ip < getNDoubleProp(); ip++ ) {
        thrust::gather( thrust::device, index.begin(), index.end(), getPtrDoubleProp( ip ), buffer.begin() );
        swapDoubleProp( ip, buffer );
    }
    buffer.clear();
    thrust::device_vector<short> buffer_short( gpu_nparts_ );
    for( int ip = 0; ip < getNShortProp(); ip++ ) {
        thrust::gather( thrust::device, index.begin(), index.end(), getPtrShortProp( ip ), buffer_short.begin() );
        swapShortProp( ip, buffer_short );
    }
    buffer_short.clear();
    if( tracked ) {
        thrust::device_vector<uint64_t> buffer_uint64( gpu_nparts_ );
        thrust::gather( thrust::device, index.begin(), index.end(), getPtrId(), buffer_uint64.begin() );
        swapId( buffer_uint64 );
        buffer_uint64.clear();
    }
}

//! Sort by cell_keys_
//! This version is asynchronous, but requires a buffer of equal size to be provided
void nvidiaParticles::sortParticleByKey( nvidiaParticles& buffer )
{
    // Make a sorting map using the cell keys (like numpy.argsort)
    thrust::device_vector<int> index( gpu_nparts_ );
    thrust::sequence( thrust::device, index.begin(), index.end() );
    thrust::sort_by_key( thrust::device, nvidia_cell_keys_.begin(), nvidia_cell_keys_.end(), index.begin() );
    
    // Sort particles using thrust::gather, according to the sorting map
    for( int ip = 0; ip < getNDoubleProp(); ip++ ) {
        thrust::gather( thrust::cuda::par_nosync, index.begin(), index.end(), getPtrDoubleProp( ip ), buffer.getPtrDoubleProp( ip ) );
    }
    for( int ip = 0; ip < getNShortProp(); ip++ ) {
        thrust::gather( thrust::cuda::par_nosync, index.begin(), index.end(), getPtrShortProp( ip ), buffer.getPtrShortProp( ip ) );
    }
    if( tracked ) {
        thrust::gather( thrust::cuda::par_nosync, index.begin(), index.end(), getPtrId(), buffer.getPtrId() );
    }
    hipDeviceSynchronize();
    
    swap( buffer );
}

int nvidiaParticles::prepareBinIndex()
{
    if( first_index.size() == 0 ) {
        // Some Particles object do not have allocated bins, we skip theses.
        return -1;
    }

    const int kGPUBinCount = parameters_->getGPUBinCount();

    if( kGPUBinCount < 0 ) {
        // Unsupported space dimension or the offloading technology is not
        // supported, dont do GPU binning.
        return -1;
    }

    // We completely ignore/discard/overwrite what's done in
    // ParticleCreator::create regarding binning.
    // NOTE: maybe ParticleCreator::create should not be doing the particle
    // binning and should only be responsible for particle initialization (pos,
    // momentum etc.).
    // We are forced to deal with first_index even though its completely
    // redundant as long as the bins are dense (no holes).

    const auto particle_count = last_index.back();

    first_index.resize( 1 );
    last_index.resize( kGPUBinCount );

    // By definition it should be zero, so this is a redundant assignment
    first_index.back() = 0;

    // Dont try to allocate 2 times, even if it's harmless, that would be a bug!
    SMILEI_ASSERT( !smilei::tools::gpu::HostDeviceMemoryManagement::IsHostPointerMappedOnDevice( last_index.data() ) );

    // We'll need last_index to be on the GPU.

    // TODO(Etienne M): FREE. If we have load balancing or other patch
    // creation/destruction available (which is not the case on GPU ATM),
    // we should be taking care of freeing this GPU memory.
    smilei::tools::gpu::HostDeviceMemoryManagement::DeviceAllocate( last_index );

    return 0;
}

void nvidiaParticles::setHostBinIndex()
{
    // TODO(Etienne M): You may want to inject, create etc. into a non binned
    // nvidiaParticles object (without allocated first/last_index). For now, we
    // assert it does not happen. I think a fix only requires:
    //  if( last_index.empty() ) { return; }
    //
    SMILEI_ASSERT( !last_index.empty() );

    last_index.back() = deviceSize();
    last_index[0]     = last_index.back();
}

void nvidiaParticles::naiveImportAndSortParticles( nvidiaParticles* particles_to_inject )
{
    // Erase particles that leaves this patch
    eraseLeavingParticles();

    // Inject newly arrived particles in particles_to_inject
    const size_t current_size = gpu_nparts_;
    resize( current_size + particles_to_inject->size() );
    copyParticles( particles_to_inject, current_size );
    particles_to_inject->clear();
}

extern "C"
{
    void* CreateGPUParticles( const void* parameters, const void* a_parent_patch )
    {
        return new nvidiaParticles{ *static_cast<const Params*>( parameters ),
                                    *static_cast<const Patch*>( a_parent_patch ) };
    }
}
