
#include <hip/hip_runtime.h>

// TODO(Etienne M): The makefile does not recognise this file and doesn't compute
// it's dependencies. If you make a modification in one of the header this file
// includes, you must `touch` this file. IF you dont do that you'll have ABI/ODR
// issues (!).

#if defined( SMILEI_ACCELERATOR_GPU_OMP )

    //! Simple switch to jump between the reference (omp) implementation and the
    //! hip one.
    //! NOTE: If you wanna use the OMP version, you must rename this file to
    //! .cpp instead of .cu for the HIP. The preprocessor and the Smilei
    //! makefile will take care of the rest.
    //!
    #if defined( __HIP__ )
    // HIP compiler support enabled (for .cu files)
    #else
        #define PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLEMENTATION 1
    #endif

    #if defined( PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLEMENTATION )
        #include <cmath>

        #include "Tools.h"
    #else
        #include <hip/hip_runtime.h>

        #include "Params.h"
        #include "gpu.h"
    #endif

    #if defined( PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLEMENTATION )

namespace naive {

    static inline void
    currentDepositionKernel( double *__restrict__ Jx,
                             double *__restrict__ Jy,
                             double *__restrict__ Jz,
                             int Jx_size,
                             int Jy_size,
                             int Jz_size,
                             const double *__restrict__ device_particle_position_x,
                             const double *__restrict__ device_particle_position_y,
                             const double *__restrict__ device_particle_momentum_z,
                             const short *__restrict__ device_particle_charge,
                             const double *__restrict__ device_particle_weight,
                             const int *__restrict__ host_bin_index,
                             unsigned int,
                             unsigned int,
                             const double *__restrict__ invgf_,
                             const int *__restrict__ iold_,
                             const double *__restrict__ deltaold_,
                             double inv_cell_volume,
                             double dx_inv,
                             double dy_inv,
                             double dx_ov_dt,
                             double dy_ov_dt,
                             int    i_domain_begin,
                             int    j_domain_begin,
                             int    nprimy,
                             int    pxr )
    {
        // The OMP implementation is NOT bin aware. As per the precondition on
        // host_bin_index, index zero always contains the number of particles.
        // See nvidiaParticles::prepareBinIndex / setHostBinIndex.
        const unsigned int bin_count      = 1;
        const int          particle_count = host_bin_index[bin_count - 1];

            // // Arrays used for the Esirkepov projection method
            // static constexpr bool kAutoDeviceFree = true;
            // const std::size_t     kTmpArraySize   = particle_count * 5;

            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sx0_buffer{ kTmpArraySize };
            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sx1_buffer{ kTmpArraySize };
            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sy0_buffer{ kTmpArraySize };
            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sy1_buffer{ kTmpArraySize };
            // // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> DSx_buffer{ kTmpArraySize };
            // // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> DSy_buffer{ kTmpArraySize };

            // double *const __restrict__ Sx0_buffer_data = Sx0_buffer.data();
            // double *const __restrict__ Sx1_buffer_data = Sx1_buffer.data();
            // double *const __restrict__ Sy0_buffer_data = Sy0_buffer.data();
            // double *const __restrict__ Sy1_buffer_data = Sy1_buffer.data();
            // // double *const __restrict__ DSx_buffer_data = DSx_buffer.data();
            // // double *const __restrict__ DSy_buffer_data = DSy_buffer.data();

        #pragma omp target     is_device_ptr /* map */ ( /* to: */                                            \
                                                     device_particle_position_x /* [0:particle_count] */, \
                                                     device_particle_position_y /* [0:particle_count] */, \
                                                     device_particle_momentum_z /* [0:particle_count] */, \
                                                     device_particle_charge /* [0:particle_count] */,     \
                                                     device_particle_weight /* [0:particle_count] */ )
        #pragma omp teams thread_limit( 64 )
        #pragma omp distribute parallel for
        for( int particle_index = 0; particle_index < particle_count; ++particle_index ) {
            const double invgf                        = invgf_[particle_index];
            const int *const __restrict__ iold        = &iold_[particle_index];
            const double *const __restrict__ deltaold = &deltaold_[particle_index];

            double Sx0[5];
            double Sx1[5];
            double Sy0[5];
            double Sy1[5];
            // double DSx[5];
            // double DSy[5];

            // double *const __restrict__ Sx0 = Sx0_buffer_data + 5 * ( particle_index - 0 );
            // double *const __restrict__ Sx1 = Sx1_buffer_data + 5 * ( particle_index - 0 );
            // double *const __restrict__ Sy0 = Sy0_buffer_data + 5 * ( particle_index - 0 );
            // double *const __restrict__ Sy1 = Sy1_buffer_data + 5 * ( particle_index - 0 );
            // // double *const __restrict__ DSx = DSx_buffer_data + 5 * ( particle_index - 0 );
            // // double *const __restrict__ DSy = DSy_buffer_data + 5 * ( particle_index - 0 );

            // Variable declaration & initialization
            // Esirkepov's paper: https://arxiv.org/pdf/physics/9901047.pdf

            // Locate the particle on the primal grid at former time-step & calculate coeff. S0
            {
                const double delta  = deltaold[0 * particle_count];
                const double delta2 = delta * delta;
                Sx0[0]              = 0.0;
                Sx0[1]              = 0.5 * ( delta2 - delta + 0.25 );
                Sx0[2]              = 0.75 - delta2;
                Sx0[3]              = 0.5 * ( delta2 + delta + 0.25 );
                Sx0[4]              = 0.0;
            }
            {
                const double delta  = deltaold[1 * particle_count];
                const double delta2 = delta * delta;
                Sy0[0]              = 0.0;
                Sy0[1]              = 0.5 * ( delta2 - delta + 0.25 );
                Sy0[2]              = 0.75 - delta2;
                Sy0[3]              = 0.5 * ( delta2 + delta + 0.25 );
                Sy0[4]              = 0.0;
            }

            // Locate the particle on the primal grid at current time-step & calculate coeff. S1
            {
                const double xpn      = device_particle_position_x[particle_index] * dx_inv;
                const int    ip       = std::round( xpn );
                const int    ipo      = iold[0 * particle_count];
                const int    ip_m_ipo = ip - ipo - i_domain_begin;
                const double delta    = xpn - static_cast<double>( ip );
                const double delta2   = delta * delta;

                Sx1[0] = 0.0;
                Sx1[1] = 0.0;
                // Sx1[2] = 0.0; // Always set below
                Sx1[3] = 0.0;
                Sx1[4] = 0.0;

                Sx1[ip_m_ipo + 1] = 0.5 * ( delta2 - delta + 0.25 );
                Sx1[ip_m_ipo + 2] = 0.75 - delta2;
                Sx1[ip_m_ipo + 3] = 0.5 * ( delta2 + delta + 0.25 );
            }
            {
                const double ypn      = device_particle_position_y[particle_index] * dy_inv;
                const int    jp       = std::round( ypn );
                const int    jpo      = iold[1 * particle_count];
                const int    jp_m_jpo = jp - jpo - j_domain_begin;
                const double delta    = ypn - static_cast<double>( jp );
                const double delta2   = delta * delta;

                Sy1[0] = 0.0;
                Sy1[1] = 0.0;
                // Sy1[2] = 0.0; // Always set below
                Sy1[3] = 0.0;
                Sy1[4] = 0.0;

                Sy1[jp_m_jpo + 1] = 0.5 * ( delta2 - delta + 0.25 );
                Sy1[jp_m_jpo + 2] = 0.75 - delta2;
                Sy1[jp_m_jpo + 3] = 0.5 * ( delta2 + delta + 0.25 );
            }

            // DSx[0] = Sx1[0] - Sx0[0];
            // DSx[1] = Sx1[1] - Sx0[1];
            // DSx[2] = Sx1[2] - Sx0[2];
            // DSx[3] = Sx1[3] - Sx0[3];
            // DSx[4] = Sx1[4] - Sx0[4];

            // DSy[0] = Sy1[0] - Sy0[0];
            // DSy[1] = Sy1[1] - Sy0[1];
            // DSy[2] = Sy1[2] - Sy0[2];
            // DSy[3] = Sy1[3] - Sy0[3];
            // DSy[4] = Sy1[4] - Sy0[4];
            // }

            // // Charge deposition on the grid

            // for( int particle_index = 0; particle_index < particle_count; ++particle_index ) {
            //     const double invgf                        = invgf_[particle_index];
            //     const int *const __restrict__ iold        = &iold_[particle_index];
            //     const double *const __restrict__ deltaold = &deltaold_[particle_index];

            //     double *const __restrict__ Sx0 = Sx0_buffer_data + 5 * ( particle_index - 0 );
            //     double *const __restrict__ Sx1 = Sx1_buffer_data + 5 * ( particle_index - 0 );
            //     double *const __restrict__ Sy0 = Sy0_buffer_data + 5 * ( particle_index - 0 );
            //     double *const __restrict__ Sy1 = Sy1_buffer_data + 5 * ( particle_index - 0 );
            //     // double *const __restrict__ DSx = DSx_buffer_data + 5 * ( particle_index - 0 );
            //     // double *const __restrict__ DSy = DSy_buffer_data + 5 * ( particle_index - 0 );

            // (x,y,z) components of the current density for the macro-particle
            const double charge_weight = inv_cell_volume * static_cast<double>( device_particle_charge[particle_index] ) * device_particle_weight[particle_index];
            const double crx_p         = charge_weight * dx_ov_dt;
            const double cry_p         = charge_weight * dy_ov_dt;
            const double crz_p         = charge_weight * ( 1.0 / 3.0 ) * device_particle_momentum_z[particle_index] * invgf;

            // This is the particle position as grid index
            // This minus 2 come from the order 2 scheme, based on a 5 points stencil from -2 to +2.
            const int ipo = iold[0 * particle_count] - 2;
            const int jpo = iold[1 * particle_count] - 2;

            for( unsigned int i = 0; i < 1; ++i ) {
                const int iloc = ( i + ipo ) * nprimy + jpo;
                    /* Jx[iloc] += tmpJx[0]; */
        #pragma omp atomic update
                Jz[iloc] += crz_p * ( Sy1[0] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) );
                double tmp = 0.0;
                for( unsigned int j = 1; j < 5; j++ ) {
                    tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + 0.5 * ( Sx1[i] - Sx0[i] ) );
        #pragma omp atomic update
                    Jy[iloc + j + pxr * ( /* i + */ ipo )] += tmp;
        #pragma omp atomic update
                    Jz[iloc + j] += crz_p * ( Sy0[j] * ( 0.5 * Sx1[i] /* + Sx0[i] */ ) +
                                              Sy1[j] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) );
                }
            }

            double tmpJx[5]{};

            for( unsigned int i = 1; i < 5; ++i ) {
                const int iloc = ( i + ipo ) * nprimy + jpo;
                tmpJx[0] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( 0.5 * ( Sy1[0] - Sy0[0] ) );
        #pragma omp atomic update
                Jx[iloc] += tmpJx[0];
        #pragma omp atomic update
                Jz[iloc] += crz_p * ( Sy1[0] * ( 0.5 * Sx0[i] + Sx1[i] ) );
                double tmp = 0.0;
                for( unsigned int j = 1; j < 5; ++j ) {
                    tmpJx[j] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( Sy0[j] + 0.5 * ( Sy1[j] - Sy0[j] ) );
        #pragma omp atomic update
                    Jx[iloc + j] += tmpJx[j];
                    tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + 0.5 * ( Sx1[i] - Sx0[i] ) );
        #pragma omp atomic update
                    Jy[iloc + j + pxr * ( i + ipo )] += tmp;

        #pragma omp atomic update
                    Jz[iloc + j] += crz_p * ( Sy0[j] * ( 0.5 * Sx1[i] + Sx0[i] ) +
                                              Sy1[j] * ( 0.5 * Sx0[i] + Sx1[i] ) );
                }
            }
        }
    }
} // namespace naive

    #else

namespace hip {
    namespace detail {
        void checkErrors( ::hipError_t an_error_code,
                          const char  *file_name,
                          int          line )
        {
            if( an_error_code != ::hipError_t::hipSuccess ) {
                std::cout << "HIP error at " << file_name << ":" << line
                          << " -> " << ::hipGetErrorString( an_error_code );
                std::exit( EXIT_FAILURE );
            }
        }
    } // namespace detail

        #define checkHIPErrors( an_expression )                           \
            do {                                                          \
                detail::checkErrors( an_expression, __FILE__, __LINE__ ); \
            } while( 0 )

    namespace kernel {
        namespace atomic {
            namespace LDS {
                __device__ void
                AddNoReturn( float *a_pointer, float a_value )
                {
                    ::atomicAdd( a_pointer, a_value );
                }

                __device__ void
                AddNoReturn( double *a_pointer, double a_value )
                {
                    ::atomicAdd( a_pointer, a_value );
                }
            } // namespace LDS

            namespace GDS {
                __device__ void
                AddNoReturn( float *a_pointer, float a_value )
                {
        #pragma clang diagnostic push
        #pragma clang diagnostic ignored "-Wdeprecated-declarations"
                    ::atomicAddNoRet( a_pointer, a_value );
        #pragma clang diagnostic pop
                }

                __device__ void
                AddNoReturn( double *a_pointer, double a_value )
                {
        #if defined( __gfx90a__ )
                    ::unsafeAtomicAdd( a_pointer, a_value );
        #else
                    ::atomicAdd( a_pointer, a_value );
        #endif
                }
            } // namespace GDS
        }     // namespace atomic

        template <typename ComputeFloat,
                  typename ReductionFloat>
        __global__ void
        // __launch_bounds__(128, 4)
        depositForAllCurrentDimensions( double *__restrict__ device_Jx,
                                        double *__restrict__ device_Jy,
                                        double *__restrict__ device_Jz,
                                        int Jx_size,
                                        int Jy_size,
                                        int Jz_size,
                                        const double *__restrict__ device_particle_position_x,
                                        const double *__restrict__ device_particle_position_y,
                                        const double *__restrict__ device_particle_momentum_z,
                                        const short *__restrict__ device_particle_charge,
                                        const double *__restrict__ device_particle_weight,
                                        const int *__restrict__ device_bin_index,
                                        const double *__restrict__ device_invgf_,
                                        const int *__restrict__ device_iold_,
                                        const double *__restrict__ device_deltaold_,
                                        ComputeFloat inv_cell_volume,
                                        ComputeFloat dx_inv,
                                        ComputeFloat dy_inv,
                                        ComputeFloat dx_ov_dt,
                                        ComputeFloat dy_ov_dt,
                                        int          i_domain_begin,
                                        int          j_domain_begin,
                                        int          nprimy,
                                        int          pxr )
        {
            // TODO(Etienne M): refactor this function. Break it into smaller
            // pieces (lds init/store, coeff computation, deposition etc..)
            // TODO(Etienne M): prefer unsigned int vs int. At least the reader
            // knows the value wont be negative.
            // TODO(Etienne M): __ldg could be used to slightly improve GDS load
            // speed. This would only have an effect on Nvidia cards as this
            // operation is a no op on AMD.
            const unsigned int workgroup_size = blockDim.x;
            const unsigned int bin_count      = gridDim.x * gridDim.y;
            const unsigned int loop_stride    = workgroup_size; // This stride should enable better memory access coalescing

            const unsigned int x_cluster_coordinate          = blockIdx.x;
            const unsigned int y_cluster_coordinate          = blockIdx.y;
            const unsigned int workgroup_dedicated_bin_index = x_cluster_coordinate * gridDim.y + y_cluster_coordinate; // The indexing order is: x * ywidth * zwidth + y * zwidth + z
            const unsigned int thread_index_offset           = threadIdx.x;

            // The unit is the cell
            const unsigned int global_x_scratch_space_coordinate_offset = x_cluster_coordinate * Params::getGPUClusterWidth( 2 /* 2D */ );
            const unsigned int global_y_scratch_space_coordinate_offset = y_cluster_coordinate * Params::getGPUClusterWidth( 2 /* 2D */ );

            // NOTE: We gain from the particles not being sorted inside a
            // cluster because it reduces the bank conflicts one gets when
            // multiple threads access the same part of the shared memory. Such
            // "conflicted" accesses are serialized !
            // NOTE: We use a bit to much LDS. For Jx, the first row could be
            // discarded, for Jy we could remove the first column.

            static constexpr unsigned int kFieldScratchSpaceSize = Params::getGPUInterpolationClusterCellVolume( 2 /* 2D */, 2 /* 2nd order interpolation */ );

            // NOTE: I tried having only one cache and reusing it. Doing that
            // requires you to iterate multiple time over the particle which is
            // possible but cost more bandwidth. The speedup was ~x0.92.
            __shared__ ReductionFloat Jx_scratch_space[kFieldScratchSpaceSize];
            __shared__ ReductionFloat Jy_scratch_space[kFieldScratchSpaceSize];
            __shared__ ReductionFloat Jz_scratch_space[kFieldScratchSpaceSize];

            // Init the shared memory

            for( unsigned int field_index = thread_index_offset;
                 field_index < kFieldScratchSpaceSize;
                 field_index += workgroup_size ) {
                Jx_scratch_space[field_index] = static_cast<ReductionFloat>( 0.0 );
                Jy_scratch_space[field_index] = static_cast<ReductionFloat>( 0.0 );
                Jz_scratch_space[field_index] = static_cast<ReductionFloat>( 0.0 );
            }

            __syncthreads();

            const unsigned int particle_count = device_bin_index[bin_count - 1];

            // This workgroup has to process distance(last_particle,
            // first_particle) particles
            const unsigned int first_particle = workgroup_dedicated_bin_index == 0 ? 0 :
                                                                                     device_bin_index[workgroup_dedicated_bin_index - 1];
            const unsigned int last_particle  = device_bin_index[workgroup_dedicated_bin_index];

            for( unsigned int particle_index = first_particle + thread_index_offset;
                 particle_index < last_particle;
                 particle_index += loop_stride ) {
                const ComputeFloat invgf                  = static_cast<ComputeFloat>( device_invgf_[particle_index] );
                const int *const __restrict__ iold        = &device_iold_[particle_index];
                const double *const __restrict__ deltaold = &device_deltaold_[particle_index];

                ComputeFloat Sx0[5];
                ComputeFloat Sx1[5];
                ComputeFloat Sy0[5];
                ComputeFloat Sy1[5];
                // double DSx[5];
                // double DSy[5];

                // Variable declaration & initialization
                // Esirkepov's paper: https://arxiv.org/pdf/physics/9901047.pdf

                // Locate the particle on the primal grid at former time-step & calculate coeff. S0
                {
                    const ComputeFloat delta  = deltaold[0 * particle_count];
                    const ComputeFloat delta2 = delta * delta;

                    Sx0[0] = static_cast<ComputeFloat>( 0.0 );
                    Sx0[1] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 - delta + static_cast<ComputeFloat>( 0.25 ) );
                    Sx0[2] = static_cast<ComputeFloat>( 0.75 ) - delta2;
                    Sx0[3] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 + delta + static_cast<ComputeFloat>( 0.25 ) );
                    Sx0[4] = static_cast<ComputeFloat>( 0.0 );
                }
                {
                    const ComputeFloat delta  = deltaold[1 * particle_count];
                    const ComputeFloat delta2 = delta * delta;

                    Sy0[0] = static_cast<ComputeFloat>( 0.0 );
                    Sy0[1] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 - delta + static_cast<ComputeFloat>( 0.25 ) );
                    Sy0[2] = static_cast<ComputeFloat>( 0.75 ) - delta2;
                    Sy0[3] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 + delta + static_cast<ComputeFloat>( 0.25 ) );
                    Sy0[4] = static_cast<ComputeFloat>( 0.0 );
                }

                // Locate the particle on the primal grid at current time-step & calculate coeff. S1
                {
                    const ComputeFloat xpn = static_cast<ComputeFloat>( device_particle_position_x[particle_index] ) * dx_inv;
                    const int          ip  = std::round( xpn );
                    // const int    ip       = static_cast<int>( xpn + 0.5 ); // std::round | rounding approximation which is correct enough and faster in this case
                    const int          ipo      = iold[0 * particle_count];
                    const int          ip_m_ipo = ip - ipo - i_domain_begin;
                    const ComputeFloat delta    = xpn - static_cast<ComputeFloat>( ip );
                    const ComputeFloat delta2   = delta * delta;

                    Sx1[0] = static_cast<ComputeFloat>( 0.0 );
                    Sx1[1] = static_cast<ComputeFloat>( 0.0 );
                    // Sx1[2] = 0.0; // Always set below
                    Sx1[3] = static_cast<ComputeFloat>( 0.0 );
                    Sx1[4] = static_cast<ComputeFloat>( 0.0 );

                    Sx1[ip_m_ipo + 1] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 - delta + static_cast<ComputeFloat>( 0.25 ) );
                    Sx1[ip_m_ipo + 2] = static_cast<ComputeFloat>( 0.75 ) - delta2;
                    Sx1[ip_m_ipo + 3] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 + delta + static_cast<ComputeFloat>( 0.25 ) );
                }
                {
                    const ComputeFloat ypn = static_cast<ComputeFloat>( device_particle_position_y[particle_index] ) * dy_inv;
                    const int          jp  = std::round( ypn );
                    // const int    jp       = static_cast<int>( ypn + 0.5 ); // std::round | rounding approximation which is correct enough and faster in this case
                    const int          jpo      = iold[1 * particle_count];
                    const int          jp_m_jpo = jp - jpo - j_domain_begin;
                    const ComputeFloat delta    = ypn - static_cast<ComputeFloat>( jp );
                    const ComputeFloat delta2   = delta * delta;

                    Sy1[0] = static_cast<ComputeFloat>( 0.0 );
                    Sy1[1] = static_cast<ComputeFloat>( 0.0 );
                    // Sy1[2] = 0.0; // Always set below
                    Sy1[3] = static_cast<ComputeFloat>( 0.0 );
                    Sy1[4] = static_cast<ComputeFloat>( 0.0 );

                    Sy1[jp_m_jpo + 1] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 - delta + static_cast<ComputeFloat>( 0.25 ) );
                    Sy1[jp_m_jpo + 2] = static_cast<ComputeFloat>( 0.75 ) - delta2;
                    Sy1[jp_m_jpo + 3] = static_cast<ComputeFloat>( 0.5 ) * ( delta2 + delta + static_cast<ComputeFloat>( 0.25 ) );
                }

                // DSx[0] = Sx1[0] - Sx0[0];
                // DSx[1] = Sx1[1] - Sx0[1];
                // DSx[2] = Sx1[2] - Sx0[2];
                // DSx[3] = Sx1[3] - Sx0[3];
                // DSx[4] = Sx1[4] - Sx0[4];

                // DSy[0] = Sy1[0] - Sy0[0];
                // DSy[1] = Sy1[1] - Sy0[1];
                // DSy[2] = Sy1[2] - Sy0[2];
                // DSy[3] = Sy1[3] - Sy0[3];
                // DSy[4] = Sy1[4] - Sy0[4];

                // (x,y,z) components of the current density for the macro-particle
                const ComputeFloat charge_weight = inv_cell_volume * static_cast<ComputeFloat>( device_particle_charge[particle_index] ) * static_cast<ComputeFloat>( device_particle_weight[particle_index] );
                const ComputeFloat crx_p         = charge_weight * dx_ov_dt;
                const ComputeFloat cry_p         = charge_weight * dy_ov_dt;
                const ComputeFloat crz_p         = charge_weight * static_cast<ComputeFloat>( 1.0 / 3.0 ) * static_cast<ComputeFloat>( device_particle_momentum_z[particle_index] ) * invgf;

                // This is the particle position as grid index
                // This minus 2 come from the order 2 scheme, based on a 5 points stencil from -2 to +2.
                const int ipo = iold[0 * particle_count] -
                                2 /* Offset so we dont uses negative numbers in the loop */ -
                                global_x_scratch_space_coordinate_offset /* Offset to get cluster relative coordinates */;
                const int jpo = iold[1 * particle_count] -
                                2 /* Offset so we dont uses negative numbers in the loop */ -
                                global_y_scratch_space_coordinate_offset /* Offset to get cluster relative coordinates */;

                // Jx

                ComputeFloat tmpJx[5]{};

                for( unsigned int i = 1; i < 5; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    tmpJx[0] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( static_cast<ComputeFloat>( 0.5 ) * ( Sy1[0] - Sy0[0] ) );
                    atomic::LDS::AddNoReturn( &Jx_scratch_space[iloc], static_cast<ReductionFloat>( tmpJx[0] ) );
                    for( unsigned int j = 1; j < 5; ++j ) {
                        tmpJx[j] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( Sy0[j] + static_cast<ComputeFloat>( 0.5 ) * ( Sy1[j] - Sy0[j] ) );
                        atomic::LDS::AddNoReturn( &Jx_scratch_space[iloc + j], static_cast<ReductionFloat>( tmpJx[j] ) );
                    }
                }

                // Jy

                for( unsigned int i = 0; i < 1; ++i ) {
                    const int    iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    ComputeFloat tmp{};
                    for( unsigned int j = 1; j < 5; j++ ) {
                        tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + static_cast<ComputeFloat>( 0.5 ) * ( Sx1[i] - Sx0[i] ) );
                        atomic::LDS::AddNoReturn( &Jy_scratch_space[iloc + j], static_cast<ReductionFloat>( tmp ) );
                    }
                }

                for( unsigned int i = 1; i < 5; ++i ) {
                    const int    iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    ComputeFloat tmp{};
                    for( unsigned int j = 1; j < 5; ++j ) {
                        tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + static_cast<ComputeFloat>( 0.5 ) * ( Sx1[i] - Sx0[i] ) );
                        atomic::LDS::AddNoReturn( &Jy_scratch_space[iloc + j], static_cast<ReductionFloat>( tmp ) );
                    }
                }

                // Jz

                for( unsigned int i = 0; i < 1; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    atomic::LDS::AddNoReturn( &Jz_scratch_space[iloc], static_cast<ReductionFloat>( crz_p * ( Sy1[0] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) ) ) );
                    for( unsigned int j = 1; j < 5; j++ ) {
                        atomic::LDS::AddNoReturn( &Jz_scratch_space[iloc + j], static_cast<ReductionFloat>( crz_p * ( Sy0[j] * ( static_cast<ComputeFloat>( 0.5 ) * Sx1[i] /* + Sx0[i] */ ) +
                                                                                                                      Sy1[j] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) ) ) );
                    }
                }

                for( unsigned int i = 1; i < 5; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    atomic::LDS::AddNoReturn( &Jz_scratch_space[iloc], static_cast<ReductionFloat>( crz_p * ( Sy1[0] * ( static_cast<ComputeFloat>( 0.5 ) * Sx0[i] + Sx1[i] ) ) ) );
                    for( unsigned int j = 1; j < 5; ++j ) {
                        atomic::LDS::AddNoReturn( &Jz_scratch_space[iloc + j], static_cast<ReductionFloat>( crz_p * ( Sy0[j] * ( static_cast<ComputeFloat>( 0.5 ) * Sx1[i] + Sx0[i] ) +
                                                                                                                      Sy1[j] * ( static_cast<ComputeFloat>( 0.5 ) * Sx0[i] + Sx1[i] ) ) ) );
                    }
                }
            }

            __syncthreads();

            for( unsigned int field_index = thread_index_offset;
                 field_index < kFieldScratchSpaceSize;
                 field_index += workgroup_size ) {

                // The indexing order is: x * ywidth * zwidth + y * zwidth + z
                const unsigned int local_x_scratch_space_coordinate = field_index / Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ );
                const unsigned int local_y_scratch_space_coordinate = field_index % Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ );

                const unsigned int global_x_scratch_space_coordinate = global_x_scratch_space_coordinate_offset + local_x_scratch_space_coordinate;
                const unsigned int global_y_scratch_space_coordinate = global_y_scratch_space_coordinate_offset + local_y_scratch_space_coordinate;

                // The indexing order is: x * ywidth * zwidth + y * zwidth + z
                const unsigned int global_memory_index = global_x_scratch_space_coordinate * nprimy + global_y_scratch_space_coordinate;
                const unsigned int scratch_space_index = field_index; // local_x_scratch_space_coordinate * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + local_y_scratch_space_coordinate;

                // These atomics are basically free (very few of them).
                atomic::GDS::AddNoReturn( &device_Jx[global_memory_index], static_cast<double>( Jx_scratch_space[scratch_space_index] ) );
                atomic::GDS::AddNoReturn( &device_Jy[global_memory_index + /* We handle the FTDT/picsar */ pxr * global_x_scratch_space_coordinate], static_cast<double>( Jy_scratch_space[scratch_space_index] ) );
                atomic::GDS::AddNoReturn( &device_Jz[global_memory_index], static_cast<double>( Jz_scratch_space[scratch_space_index] ) );
            }
        }
    } // namespace kernel

    static inline void
    currentDepositionKernel( double *__restrict__ host_Jx,
                             double *__restrict__ host_Jy,
                             double *__restrict__ host_Jz,
                             int Jx_size,
                             int Jy_size,
                             int Jz_size,
                             const double *__restrict__ device_particle_position_x,
                             const double *__restrict__ device_particle_position_y,
                             const double *__restrict__ device_particle_momentum_z,
                             const short *__restrict__ device_particle_charge,
                             const double *__restrict__ device_particle_weight,
                             const int *__restrict__ host_bin_index,
                             unsigned int x_dimension_bin_count,
                             unsigned int y_dimension_bin_count,
                             const double *__restrict__ host_invgf_,
                             const int *__restrict__ host_iold_,
                             const double *__restrict__ host_deltaold_,
                             double inv_cell_volume,
                             double dx_inv,
                             double dy_inv,
                             double dx_ov_dt,
                             double dy_ov_dt,
                             int    i_domain_begin,
                             int    j_domain_begin,
                             int    nprimy,
                             int    pxr )
    {
        int device_count;
        checkHIPErrors( ::hipGetDeviceCount( &device_count ) );
        SMILEI_ASSERT( device_count == 1 );

        SMILEI_ASSERT( Params::getGPUClusterWidth( 2 /* 2D */ ) != -1 &&
                       Params::getGPUClusterGhostCellBorderWidth( 2 /* 2nd order interpolation */ ) != -1 );

        // NOTE:
        // This cluster is very strongly bound by atomic operations in LDS (shared memory)
        // TODO(Etienne M): Find a way to lessen the atomic usage

        const ::dim3 kGridDimensionInBlock{ static_cast<uint32_t>( x_dimension_bin_count ), static_cast<uint32_t>( y_dimension_bin_count ), 1 };
        // On an MI100:
        // 448 for F32 and 4x4 cluster width | past 128, the block size does not matter, we are atomic bound anyway
        // 128 for F64 and 4x4 cluster width | atomic bound
        const ::dim3 kBlockDimensionInWorkItem{ 128, 1, 1 };

        // On MI100, using float for reduction reduces the amount of bank 
        // conflict and allows the compiler to generate better instruction.
        // The relative error is ~10^13 compared to pure double operations but
        // is x1.3 times faster.

        using ComputeFloat   = double;
        using ReductionFloat = double;

        auto KernelFunction = kernel::depositForAllCurrentDimensions<ComputeFloat, ReductionFloat>;

        hipLaunchKernelGGL( KernelFunction,
                            kGridDimensionInBlock,
                            kBlockDimensionInWorkItem,
                            0, // Shared memory
                            0, // Stream
                            // Kernel arguments
                            smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( host_Jx ),
                            smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( host_Jy ),
                            smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( host_Jz ),
                            Jx_size, Jy_size, Jz_size,
                            device_particle_position_x,
                            device_particle_position_y,
                            device_particle_momentum_z,
                            device_particle_charge,
                            device_particle_weight,
                            smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( host_bin_index ),
                            smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( host_invgf_ ),
                            smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( host_iold_ ),
                            smilei::tools::gpu::HostDeviceMemoryManagement::GetDevicePointer( host_deltaold_ ),
                            inv_cell_volume,
                            dx_inv, dy_inv,
                            dx_ov_dt, dy_ov_dt,
                            i_domain_begin, j_domain_begin,
                            nprimy,
                            pxr );

        checkHIPErrors( ::hipDeviceSynchronize() );
    }

} // namespace hip

    #endif

//! Project global current densities (EMfields->Jx_/Jy_/Jz_)
//!
extern "C" void
currentDepositionKernel( double *__restrict__ host_Jx,
                         double *__restrict__ host_Jy,
                         double *__restrict__ host_Jz,
                         int Jx_size,
                         int Jy_size,
                         int Jz_size,
                         const double *__restrict__ device_particle_position_x,
                         const double *__restrict__ device_particle_position_y,
                         const double *__restrict__ device_particle_momentum_z,
                         const short *__restrict__ device_particle_charge,
                         const double *__restrict__ device_particle_weight,
                         const int *__restrict__ host_bin_index,
                         unsigned int x_dimension_bin_count,
                         unsigned int y_dimension_bin_count,
                         const double *__restrict__ host_invgf_,
                         const int *__restrict__ host_iold_,
                         const double *__restrict__ host_deltaold_,
                         double inv_cell_volume,
                         double dx_inv,
                         double dy_inv,
                         double dx_ov_dt,
                         double dy_ov_dt,
                         int    i_domain_begin,
                         int    j_domain_begin,
                         int    nprimy,
                         int    pxr )
{
    #if defined( PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLEMENTATION )
    naive:: // the naive, OMP version serves as a reference along with the CPU version
    #else
    hip::
    #endif
        currentDepositionKernel( host_Jx, host_Jy, host_Jz,
                                 Jx_size, Jy_size, Jz_size,
                                 device_particle_position_x, device_particle_position_y,
                                 device_particle_momentum_z,
                                 device_particle_charge,
                                 device_particle_weight,
                                 host_bin_index,
                                 x_dimension_bin_count,
                                 y_dimension_bin_count,
                                 host_invgf_,
                                 host_iold_, host_deltaold_,
                                 inv_cell_volume,
                                 dx_inv, dy_inv,
                                 dx_ov_dt, dy_ov_dt,
                                 i_domain_begin, j_domain_begin,
                                 nprimy,
                                 pxr );
}

#endif
